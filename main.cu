#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>

#include "support.h"

int main(int argc, char* argv[])
{
    Timer timer;
    startTime(&timer);

    int c;
    int line_count = 0;
    FILE *file;
    hipError_t cuda_ret;

    const char* filename;

    if(argc == 1) {
        filename = "cemetery.csv";
    } else if(argc == 2) {
        filename = argv[1];
    }

    printf("\nLoading file...");

    file = fopen(filename, "r");
    if (file) {
        while ((c = getc(file)) != EOF)
            if(c == '\n') line_count++;
    }

    float *lats_h = (float*) malloc(sizeof(float)*line_count);
    float *lons_h = (float*) malloc(sizeof(float)*line_count);

    float *lats_d;
    float *lons_d;

    int i = 0;
    rewind(file);
    while (i < line_count) {
        fscanf(file, "%f,", &lats_h[i]);
        fscanf(file, "%f", &lons_h[i]);
        i++;
    }

    fclose(file);

    printf("\n%d points loaded. ", line_count);
    stopTime(&timer); printf("%f s", elapsedTime(timer));

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMalloc((float**)&lats_d, line_count * sizeof(float));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    cuda_ret = hipMalloc((float**)&lons_d, line_count * sizeof(float));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    hipDeviceSynchronize();
    
    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMemcpy(lats_d, lats_h, line_count * sizeof(float),
        hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device");

    cuda_ret = hipMemcpy(lons_d, lons_h, line_count * sizeof(float),
        hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to set device memory");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    hipFree(lats_d); hipFree(lons_d);
    free(lats_h); free(lons_h);

    return 0;
}
