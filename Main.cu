#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>

#include "support.h"

int main(int argc, char* argv[])
{
    Timer timer;
    startTime(&timer);

    int c;
    int line_count = 0;
    FILE *file;
    hipError_t cuda_ret;

    printf("\nLoading file...");

    char* filename = atoi(argv[1]);

    file = fopen(filename, "r");
    if (file) {
        while ((c = getc(file)) != EOF)
            if(c == '\n') line_count++;
    }

    float *lats_h = (float*) malloc(sizeof(float)*line_count);
    float *lons_h = (float*) malloc(sizeof(float)*line_count);

    int i = 0;
    rewind(file);
    while (i < line_count) {
        fscanf(file, "%f,", &lats[i]);
        fscanf(file, "%f", &lons[i]);
        i++;
    }

    fclose(file);

    printf("\n%d points loaded. ", line_count);
    stopTime(&timer); printf("%f s", elapsedTime(timer));

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMalloc((void**)&lats_d, line_count * sizeof(unsigned float));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    cuda_ret = hipMalloc((void**)&lons_d, line_count * sizeof(unsigned float));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    hipDeviceSynchronize();
    
    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMemcpy(lats_d, lats_h, line_count * sizeof(unsigned float),
        hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device");

    cuda_ret = hipMemset(lons_d, lons_h, line_count * sizeof(unsigned float));
    if(cuda_ret != hipSuccess) FATAL("Unable to set device memory");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    hipFree(lats_d); hipFree(lons_d);
    free(lats_h); free(lons_h);

    return 0;
}